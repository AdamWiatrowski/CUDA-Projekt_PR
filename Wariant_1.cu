#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 1026 // rozmiar macierzy;
#define R 33 // promien sumowania
#define K 1 // ilosc elementow obliczana przez kazdy watek

#define OUT_SIZE (N - 2 * R) // rozmiar tablicy wyjsciowej
#define BLOCK_SIZE 32 // rozmiar bloku
#define GRID_SIZE ((OUT_SIZE/BLOCK_SIZE)*(OUT_SIZE/(K*BLOCK_SIZE)))
#define SIZE_X (BLOCK_SIZE + 2*R)
#define SIZE_Y (K*BLOCK_SIZE + 2*R)

__global__ void calculate(int* out, int* tab, int N_k, int R_k, int K_k) {

    __shared__ int shared_tab[SIZE_X][SIZE_Y];

    int row = threadIdx.y;
    int col = K_k * threadIdx.x;

    // zmienne potrzebne do przesunięcia
    int in_row = (OUT_SIZE / (K_k * BLOCK_SIZE));
    int x = blockIdx.x;
    int shift_col = (x % (OUT_SIZE / (K_k * BLOCK_SIZE))) * (K_k * BLOCK_SIZE);
    int shift_row = floorf(x / in_row) * BLOCK_SIZE;

    // kopiowanie z pamieci globalnej do pamieci wspoldzielonej (jeden watek)
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int row = 0; row < SIZE_X; row++) {
            for (int col = 0; col < SIZE_Y; col++) {
                // pobieramy odpowiednie dane w z przesunieciem
                shared_tab[row][col] = tab[(row + shift_row) * N_k + col + shift_col];
            }
        }
    }

    // synchronizacja watkow
    __syncthreads();


    // obliczenia
    for (int offset = 0; offset < K_k; offset++) {
        int i = R_k + row;
        int j = R_k + col + offset;
        int sum = 0;
        for (int k = i - R_k; k <= i + R_k; k++) {
            for (int l = j - R_k; l <= j + R_k; l++) {
                sum += shared_tab[k][l];
            }
        }
        out[(row + shift_row) * OUT_SIZE + (col + shift_col + offset)] = sum;

    }

}

// wyliczanie wynikow na cpu = sekwencyjnie;
void calculate_cpu(int* out, int* tab) {
    for (int row = 0; row < OUT_SIZE; row++) {
        for (int col = 0; col < OUT_SIZE; col++) {
            int i = R + row;
            int j = R + col;
            int sum = 0;
            for (int k = i - R; k <= i + R; k++) {
                for (int l = j - R; l <= j + R; l++) {
                    sum += tab[k * N + l];
                }
            }
            out[row * OUT_SIZE + col] = sum;
        }
    }
}

// porownanie dwoch macierzy
void compare(int* a, int* b, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (a[i * size + j] != b[i * size + j]) {
                printf("Equal = False\n");
                return;
            }
        }
    }
    printf("Equal = True\n");
}

// wypelnienie macierzy losowymi danymi
void randomize(int* array, int size) {
    srand(time(0));
    for (int i = 0; i < size * size; i++) {
        array[i] = rand() % 3 + 1;
    }
}

// wypisanie tablicy
void printArray(int* array, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%d ", array[i * size + j]);
        }
        printf("\n");
    }
}

int main() {

    int N_kernel = N;
    int R_kernel = R;
    int K_kernel = K;


    static int tab[N][N];
    static int out[OUT_SIZE][OUT_SIZE];
    static int out_cpu[OUT_SIZE][OUT_SIZE];

    randomize((int*)tab, N);

    //printArray((int*)tab, N);

    int* dev_tab, * dev_out;


    hipMalloc((void**)&dev_tab, N * N * sizeof(int));
    hipMalloc((void**)&dev_out, OUT_SIZE * OUT_SIZE * sizeof(int));


    // synchroniczne kopiowanie z CPU do GPU - wariant 3.
    hipMemcpy(dev_tab, tab, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    //dim3 dimGrid(GRID_SIZE);

    calculate << <GRID_SIZE, dimBlock >> > (dev_out, dev_tab, N_kernel, R_kernel, K_kernel);


    // synchroniczne kopiowanie z GPU do CPU - wariant 3.
    hipMemcpy(out, dev_out, OUT_SIZE * OUT_SIZE * sizeof(int), hipMemcpyDeviceToHost);


    //printf("\nGPU\n");
    //printArray((int*)out, OUT_SIZE);

    // Obliczenia na CPU
    calculate_cpu((int*)out_cpu, (int*)tab);

    //printf("\nCPU\n");
    //printArray((int*)out_cpu, OUT_SIZE);

    // porownanie tablic
    compare((int*)out, (int*)out_cpu, OUT_SIZE);

    hipFree(dev_tab);
    hipFree(dev_out);

    return 0;
}